#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>


// Device run
// Matrix multiplication
// Same GEMM Configuration

__global__ void matMul(int transA, int transB,
		int M, int N, int K, 
		float ALPHA, float *A, int lda, 
		float *B, int ldb, float BETA, 
		float *C, int ldc)
{
    int n =  blockIdx.y * blockDim.y + threadIdx.y;
    int m =  blockIdx.x * blockDim.x + threadIdx.x;
 
    if( n < N && m < M );
    {
        float sum = 0;
        for (int k = 0; k < K; k++)
        {
            sum += B[n * K + k] * A[m * K + k];
        }
        C[m * N + n] = ALPHA * sum;
        
    }
}

// for use Cuda
// cuda is C++ base
// so for "C" use
// write extern "C"

extern "C" void gemmCuda(int transA, int transB, 
		int M, int N, int K, 
		float ALPHA, float *A, int lda, 
		float *B, int ldb, float BETA, 
		float *C, int ldc)
{	// for check time
	// overhead	   : move data
	//				 host to device and device to host
	// computation : metmul
	// send		 -> host to device
	// send_host -> device to host
	// kernel	 -> metmul
	hipEvent_t  send_start, send_stop, kernel_start, kernel_stop,
	send_host_start, send_host_stop;
	float foncTime, send_time, send_host_time = 0;


    float *temp_A, *temp_B, *temp_C;

    hipMalloc((void **)&temp_A, M * K * sizeof(float));
    hipMalloc((void **)&temp_B, N * K * sizeof(float));
    hipMalloc((void **)&temp_C, M * N * sizeof(float));

	// send time overhead
	hipEventCreate(&send_start);
	hipEventCreate(&send_stop);
	hipEventRecord(send_start);

	//send input data from host to device
    hipMemcpy(temp_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(temp_B, B, N * K * sizeof(float), hipMemcpyHostToDevice);
 	
	hipEventRecord(send_stop);
	hipEventSynchronize(send_stop);
	
	hipEventElapsedTime(&send_time, send_start, send_stop);
	fprintf(stderr, "sned input data from host to device  Elapsed Time = %.6f millisec\n",
			send_time);

	hipEventDestroy(send_start);
	hipEventDestroy(send_stop);
	//fin check send time overhead

	dim3 blocksPerGrid(M/16,N/16); // 1 , 256/16
    dim3 threadsPerBlock(16,16);   // 16, 16 

	// Why did 16
	// 16 -> output chenel


	//matmul time check
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);

	matMul<<<blocksPerGrid, threadsPerBlock>>>(0, 1, M, N, K, 1.0f, temp_A, K, temp_B, N, 0, temp_C, N);
	hipDeviceSynchronize();
	

	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);

	hipEventElapsedTime(&foncTime, kernel_start, kernel_stop);
	fprintf(stderr, "MatMul Elapsed Time = %.6f millisec\n", foncTime);

	hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_stop);
	//fin matmul time 

	//send time overhead
	//send result from device to host

	memset(C, 0, M * N * sizeof(float));

	// send time overhead
	hipEventCreate(&send_host_start);
	hipEventCreate(&send_host_stop);
	hipEventRecord(send_host_start);

 
	hipMemcpy(C, temp_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(send_host_stop);
	hipEventSynchronize(send_host_stop);

	hipEventElapsedTime(&send_host_time, send_host_start, send_host_stop);
	fprintf(stderr, "sned result from device to host Elapsed Time = %.6f millisec\n",
			send_host_time);

	hipEventDestroy(send_host_start);
	hipEventDestroy(send_host_stop);
	//fin check send time overhead

	hipFree(temp_A);
    hipFree(temp_B);
    hipFree(temp_C);

}



