#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>
#include "math.h"

__global__ void matMul(int transA, int transB,
		int M, int N, int K, 
		float ALPHA, float *A, int lda, 
		float *B, int ldb, float BETA, 
		float *C, int ldc)
{
    int n =  blockIdx.y * blockDim.y + threadIdx.y;
    int m =  blockIdx.x * blockDim.x + threadIdx.x;
 
    if( n < N && m < M);
    {
        float sum = 0;
        for (int k = 0; k < K; k++)
        {
            sum += B[n * K + k] * A[m * K + k];
        }
        C[n * M + m] = ALPHA * sum;
        
    }
}

extern "C" void gemmCuda(int transA, int transB, 
		int M, int N, int K, 
		float ALPHA, float *A, int lda, 
		float *B, int ldb, float BETA, 
		float *C, int ldc)
{


    float *temp_A, *temp_B, *temp_C;

    hipMalloc((void **)&temp_A, M * K * sizeof(float));
    hipMalloc((void **)&temp_B, N * K * sizeof(float));
    hipMalloc((void **)&temp_C, M * N * sizeof(float));



	//send input data from host to device
    hipMemcpy(temp_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(temp_B, B, N * K * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocksPerGrid(M/N,N/N);
    dim3 threadsPerBlock(N,N);
	

	matMul<<<blocksPerGrid, threadsPerBlock>>>(0, 1, M, N, K, 1.0f, temp_A, K, temp_B, N, 0, temp_C, N);
	hipDeviceSynchronize();
	
	
 
	hipMemcpy(C, temp_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(temp_A);
    hipFree(temp_B);
    hipFree(temp_C);

}



